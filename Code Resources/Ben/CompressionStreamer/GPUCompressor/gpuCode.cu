#include "hip/hip_runtime.h"
/*
    Predictive 32-bit IEEE 754 floating point data compressor
    Copyright (C) 2013  benjamin bennahugo@aol.com

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "gpuCode.h"

uint32_t * _gpuCompressorIV = NULL;
uint64_t _gpuCompressorIVLength = -1;
uint32_t _gpuAccumCompressedDataSize = 0;
uint32_t * _gpuDecompressorIV = NULL;
uint64_t _gpuDecompressorIVLength = -1;
double _gpuCompressorAccumulatedTime = 0;
double _gpuDecompressorAccumulatedTime = 0;
uint32_t _gpuAccumDecompressedDataSize = 0;
const uint8_t gpuStorageIndiceCapacity = 8*sizeof(uint32_t);
const uint8_t gpuBitCountForRepresentation = 2;
uint32_t gpuBlockSize = 0; //set by initCUDA

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

void gpuCode::initCUDA(){
  int deviceCount, device;
    int gpuDeviceCount = 0;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t * properties = new hipDeviceProp_t[deviceCount];
    
    if (cudaResultCode != hipSuccess) 
        deviceCount = 0;
    /* machines with no GPUs can still report one emulation device */
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties[device], device);
        if (properties[device].major != 9999) /* 9999 means emulation only */
            ++gpuDeviceCount;
    }
    std::cout << gpuDeviceCount << " GPU CUDA device(s) found" << std::endl;
    if (gpuDeviceCount == 0){
	std::cout << "FATAL: NO CUDA CAPABLE CARDS ON THIS SYSTEM" << std::endl;
        exit(1);
    }
    hipSetDevice(0);
    gpuBlockSize = 256;
    size_t free = 0;
    size_t total = 0;
    hipMemGetInfo(&free,&total);
    std::cout << "Total GPU Memory on card: " << total/1024/1024 << " MB" << std::endl;
    std::cout << "Total GPU Memory available: " << free/1024/1024 << " MB" << std::endl;
    delete[] properties;
}
void gpuCode::releaseCard(){
  CUDA_CHECK_RETURN(hipDeviceReset());
}
/*
 * Inits the compressor
 * @params iv the first dataframe that serves as a basis for the compresson of further dataframes
 * @params ivlength the length of the iv vector
 * @throws invalidInitializationException if the IV is empty
 */
void gpuCode::compressor::initCompressor(const float* iv, uint64_t ivLength){
  if (ivLength < 1)
    throw invalidInitializationException();
  uint32_t numBlocks = (ivLength / gpuBlockSize) + (ivLength%gpuBlockSize != 0); //+1 iff there is remaining elements after number of completely fulled blocks
  if (_gpuCompressorIV != NULL)
    CUDA_CHECK_RETURN(hipFree(_gpuCompressorIV));
  CUDA_CHECK_RETURN(hipMalloc((void**) &_gpuCompressorIV, sizeof(uint32_t) * numBlocks * gpuBlockSize));
  CUDA_CHECK_RETURN(hipMemset(_gpuCompressorIV,0,sizeof(uint32_t) * numBlocks * gpuBlockSize)); //ensure padding is set to zero on the device
  CUDA_CHECK_RETURN(hipMemcpy(_gpuCompressorIV, iv, ivLength*sizeof(float), hipMemcpyHostToDevice));
  _gpuCompressorIVLength = ivLength;
  _gpuCompressorAccumulatedTime = 0;
  _gpuAccumCompressedDataSize = ivLength+1;
}

/*
 * Releases resources held by the compressor
 */
void gpuCode::compressor::releaseResources(){
  if (_gpuCompressorIV != NULL){
    CUDA_CHECK_RETURN(hipFree((void*) _gpuCompressorIV));
    _gpuCompressorIV = NULL;
    _gpuCompressorIVLength = -1;
  }
}

/*
 * Gets the accumulated time since compressor initialization
 */
double gpuCode::compressor::getAccumulatedRunTimeSinceInit(){
  return _gpuCompressorAccumulatedTime;
}

/*
 * Gets the accumulated size needed to store the data since initialization of the compressor 
 * (this can be used to compute the compression ratio)
 */
uint32_t gpuCode::compressor::getAccumulatedCompressedDataSize(){
  return _gpuAccumCompressedDataSize;
}

const uint32_t BYTESPERINTMIN1_2 = sizeof(uint32_t) * sizeof(uint8_t) - 1;
inline int32_t imax_2( int32_t x, int32_t y )
{
    return x - ((x - y) & ((x - y) >> (BYTESPERINTMIN1_2)));
}
inline int32_t imin_2( int32_t x, int32_t y )
{
    return y ^ ((x ^ y) & -(x < y)); // min(x, y)
}

__device__ uint32_t storePrefixStream(const uint32_t * iv, uint32_t elementCount, uint32_t chunkSize, 
			uint32_t ** prefixStore, uint32_t ** residualStore, 
			uint32_t lowerBound,uint32_t blockThreadId,uint32_t index,uint32_t dataElement,
			uint32_t bankOffset){
    extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement  
    
    uint32_t element = 0;
    uint32_t lshiftAmountPrefixes = gpuStorageIndiceCapacity - gpuBitCountForRepresentation;
    if ((index < elementCount)){
        //save the prefixes:
	uint32_t ivElement = iv[index];
        element = ivElement ^ dataElement;
	uint32_t prefix0 = min(3,__clz(element) >> 3);
        uint32_t iTimesgpuBitCountForRepresentation = blockThreadId*gpuBitCountForRepresentation;
        uint32_t startingIndex = (iTimesgpuBitCountForRepresentation) >> 5;
        uint32_t rshiftAmount = (iTimesgpuBitCountForRepresentation) % gpuStorageIndiceCapacity;
	atomicOr(prefixStore[blockIdx.x] + startingIndex,
		 ((prefix0 << lshiftAmountPrefixes) >> rshiftAmount)); //according to the cuda developer guide this will compute the or and store it back to the same address
	//store a copy of the orignal count at an 2* BLOCK SIZE offset in shared memory so that we can get the scan values and originals later!
	uint32_t countIndexN = blockThreadId+bankOffset;
        counts[countIndexN] = ((sizeof(uint32_t)-prefix0) << 3);
	counts[(chunkSize<<1)+countIndexN] = counts[countIndexN];
    }
    return element;
}

/**
 * Define a macro according to Nvidia's GPU Gems 3 to offset the indexing in order to avoid bank conflicts
 * Compute Capability 	1.x: 16 banks
 * 			2.x/3.x: 32 banks
 */
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \ ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

__device__ void computeScan(uint32_t n, uint32_t blockSize) {
	extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement
	int thid = threadIdx.x; //TODO::use a more complex indexing scheme to deal with larger arrays
	int blockOffset = blockIdx.x*blockSize;
	int offset = 1;
	//up-sweep:
	for (int d = blockSize>>1; d > 0; d >>= 1)                    // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d){
			int thidTimes2 = (thid<<1);
		    int ai = offset*(thidTimes2+1)-1;
		    int bi = offset*(thidTimes2+2)-1;
		    ai += CONFLICT_FREE_OFFSET(ai);
		    bi += CONFLICT_FREE_OFFSET(bi);
		    counts[bi] += counts[ai];
		}
		offset <<= 1;
	}
	//clear:
	if (thid==0) {
		int iMax = blockSize - 1;
		counts[iMax + CONFLICT_FREE_OFFSET(iMax)] = 0;
	}
	//down-sweep:
	for (int d = 1; d < blockSize; d <<= 1) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d){
			int thidTimes2 = (thid<<1);
			int ai = offset*(thidTimes2+1)-1;
			int bi = offset*(thidTimes2+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			uint32_t t = counts[ai];
			counts[ai] = counts[bi];
			counts[bi] += t;
		}
	}
}

__device__ uint32_t storeResidualStream(uint32_t elementCount, uint32_t chunkSize, 
			uint32_t ** prefixStore, uint32_t ** residualStore, 
			uint32_t lowerBound,uint32_t blockThreadId,uint32_t index,uint32_t xoredElement,
			uint32_t bankOffset){
  extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement  
  if ((index < elementCount)){ //exclude the last element
	//save the residuals:
        uint32_t countIndexN = blockThreadId+bankOffset;
        uint32_t accumulatedIndex = counts[countIndexN];
	uint32_t count = counts[(chunkSize<<1) + countIndexN]; //get the original count value before the prefix scan was computed
        uint32_t startingIndex = accumulatedIndex >> 5;
        uint8_t lshiftAmount = (gpuStorageIndiceCapacity - count);
        uint32_t rshiftAmount = accumulatedIndex % gpuStorageIndiceCapacity;
        uint8_t writtenBits = gpuStorageIndiceCapacity - lshiftAmount - max(rshiftAmount-lshiftAmount,0);
        atomicOr(residualStore[blockIdx.x] + startingIndex,((xoredElement << lshiftAmount) >> rshiftAmount));
        atomicOr(residualStore[blockIdx.x] + startingIndex + 1,(xoredElement << (lshiftAmount + writtenBits - 1) << 1));    
    }
}
__global__ void gpuCompressionKernel(const uint32_t * data, uint32_t * iv, uint32_t elementCount, uint32_t chunkSize, 
			uint32_t ** prefixStore, uint32_t ** residualStore, uint32_t * residualSizeStore){
      extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement
    
    //Create difference array, count used bits (up to 3 bytes of leading zeros) and save prefixes
    uint32_t lowerBound = blockIdx.x*chunkSize;
    uint32_t blockThreadId = threadIdx.x;
    //in line with the way GPU Gems 3 structures the parallel prefix sum we have to copy TWO data elements into registers
    uint32_t ai = blockThreadId;
    uint32_t bi = blockThreadId + (chunkSize>>1);
    uint32_t bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    uint32_t bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    uint32_t  blockOffsetA = lowerBound + ai;
    uint32_t  blockOffsetB = lowerBound + bi;
    uint32_t dataElementA = NULL;
    uint32_t dataElementB = NULL;
    uint32_t lastElemVal = 0;
    
    if (blockOffsetA < elementCount)
      dataElementA = data[blockOffsetA];
    if (blockOffsetB < elementCount)
      dataElementB = data[blockOffsetB];    
    //compute lzc and save the prefixes:
     uint32_t elementA = storePrefixStream(iv,elementCount,chunkSize,prefixStore,residualStore,lowerBound,ai,blockOffsetA,dataElementA,bankOffsetA);
     uint32_t elementB = storePrefixStream(iv,elementCount,chunkSize,prefixStore,residualStore,lowerBound,bi,blockOffsetB,dataElementB,bankOffsetB);
    __syncthreads();
    //compute parallel prefix sum (this method taken from GPU GEMS 3 computes 2 elements at a time):
     computeScan(elementCount,chunkSize);
     __syncthreads();
    //now save the residuals:
    storeResidualStream(elementCount,chunkSize,prefixStore,residualStore,lowerBound,ai,blockOffsetA,elementA,bankOffsetA);
    storeResidualStream(elementCount,chunkSize,prefixStore,residualStore,lowerBound,bi,blockOffsetB,elementB,bankOffsetB);
    __syncthreads();
    //calculate storage space used by residuals:
    if (blockOffsetA == elementCount-1){ //last element before end of block
       uint32_t accumulatedIndex = counts[bankOffsetA+ai] + counts[(chunkSize << 1) + bankOffsetA+ai];
       uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
                           (accumulatedIndex % gpuStorageIndiceCapacity != 0);
      //Store pointers to the current prefixes and residuals
        residualSizeStore[blockIdx.x] = sizeOfResidualArray; 
    } else if (blockOffsetB == elementCount-1){
       uint32_t accumulatedIndex = counts[bankOffsetB+bi] + counts[(chunkSize << 1) + bankOffsetB+bi];
       uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
                           (accumulatedIndex % gpuStorageIndiceCapacity != 0);
      //Store pointers to the current prefixes and residuals
        residualSizeStore[blockIdx.x] = sizeOfResidualArray; 
    } else if (blockThreadId == (chunkSize >> 1)-1){ //last thread of block
       uint32_t lastCountElemIndex = bankOffsetB+bi;
      uint32_t accumulatedIndex = counts[lastCountElemIndex] + counts[(chunkSize << 1) + lastCountElemIndex];
      uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
                          (accumulatedIndex % gpuStorageIndiceCapacity != 0);
      residualSizeStore[blockIdx.x] = sizeOfResidualArray; 
    }
     __syncthreads();
    //Copy the current data to the IV memory for the next round of compression
    if (blockOffsetA < elementCount)
      iv[blockOffsetA] = data[blockOffsetA];
    if (blockOffsetB < elementCount)
      iv[blockOffsetB] = data[blockOffsetB];
    //the prefixes and residluals will be freed by the caller
}

void gpuDecompressionKernel(uint32_t chunkSize, uint32_t dataBlockSize, 
			  uint32_t * compressedPrefixes, uint32_t * compressedResiduals,
			  uint32_t dataBlockIndex,uint32_t lowerBound) {
    uint32_t accumulatedIndex = 0;
    uint8_t lshiftAmount = (gpuStorageIndiceCapacity - gpuBitCountForRepresentation);
    for (uint32_t i = 0; i < dataBlockSize; ++i) {
	//inflate prefix
	uint32_t prefixIndex = i*gpuBitCountForRepresentation;
        uint32_t startingIndex = prefixIndex >> 5;
        uint8_t rshiftAmount = prefixIndex % gpuStorageIndiceCapacity;
        uint8_t prefix = ((compressedPrefixes[startingIndex] << rshiftAmount) >> lshiftAmount);
        uint32_t count = gpuStorageIndiceCapacity - (prefix << 3);
	//inflate its associated residual
	startingIndex = accumulatedIndex >> 5;
        uint8_t residuallshiftAmount = (gpuStorageIndiceCapacity - count);
        rshiftAmount = accumulatedIndex % gpuStorageIndiceCapacity;
        uint8_t writtenBits = gpuStorageIndiceCapacity - residuallshiftAmount - imax_2(rshiftAmount-residuallshiftAmount,0);
        register uint32_t residual = ( (compressedResiduals[startingIndex] << rshiftAmount) >> residuallshiftAmount);
        residual |= 
	  ( compressedResiduals[startingIndex+(gpuStorageIndiceCapacity - residuallshiftAmount - writtenBits > 0)] >> (residuallshiftAmount + writtenBits - 1) >> 1);
        _gpuDecompressorIV[lowerBound+i] ^= residual;
        accumulatedIndex += count;
    }
}


/*
 * Compresses a dataframe. This function will compress a dataframe in parallel and will call back with the compressed data when completed.
 * The user should save the initialization vector dataframe and the elementCount to file himself. For dataframe index > 1 the user
 * should save the compressed prefix and residual array to persistent storage within the scope of the callback function. After the
 * return of the callback function the compressed data will be deleted from memory and the pointers will no longer be valid.
 * @throws invalidInitializationException if the length of the dataframe vector does not match the length of the initialization vector
 */

void gpuCode::compressor::compressData(const float * data, uint32_t elementCount,
			  void (*callBack)(uint32_t elementCount, uint32_t * compressedResidualsIntCounts, uint32_t ** compressedResiduals,
			    uint32_t * compressedPrefixIntCounts, uint32_t ** compressedPrefixes, uint32_t chunkCount, uint32_t * chunkSizes)){
    if (_gpuCompressorIV == NULL || _gpuCompressorIVLength != elementCount)
        throw invalidInitializationException();
    uint32_t chunkSize = gpuBlockSize; 
    uint32_t numStores = elementCount/gpuBlockSize + (elementCount%gpuBlockSize != 0); //+1 iff there is remaining elements after number of completely fulled blocks
    uint32_t** residlualStore = new uint32_t*[numStores];
    uint32_t** prefixStore = new uint32_t*[numStores];
    uint32_t* residualSizesStore = new uint32_t[numStores];
    uint32_t* prefixSizesStore = new uint32_t[numStores];
    uint32_t* chunkSizes = new uint32_t[numStores];
    //alloc space for the data on the card
    uint32_t* gpuData = NULL;
    CUDA_CHECK_RETURN(hipMalloc((void**) &gpuData, sizeof(uint32_t) * numStores * gpuBlockSize));
    CUDA_CHECK_RETURN(hipMemset(gpuData,0,sizeof(uint32_t) * numStores * gpuBlockSize)); //ensure padding is set to zero on the device
    CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, elementCount*sizeof(float), hipMemcpyHostToDevice));
    //alloc space for residuals and prefixes
    uint32_t** gpuResidlualMemoryStores = NULL;
    uint32_t** gpuPrefixMemoryStores = NULL;
    uint32_t** gpuResidlualStoresWrapper = NULL;
    uint32_t** gpuPrefixStoresWrapper = NULL;
    uint32_t* gpuResidualSizesStore = NULL;
    uint32_t* gpuPrefixSizesStore = NULL;
    //create temporary stores for the residual and prefix memory:
    gpuResidlualMemoryStores = new uint32_t*[numStores];
    gpuPrefixMemoryStores = new uint32_t*[numStores];
    //create the wrappers for the residual and prefix memory stores which will be sent to the kernel:
    CUDA_CHECK_RETURN(hipMalloc((void**) &gpuResidualSizesStore, sizeof(uint32_t) * numStores));
    CUDA_CHECK_RETURN(hipMemset(gpuResidualSizesStore,0,sizeof(uint32_t) * numStores)); 
    CUDA_CHECK_RETURN(hipMalloc((void**) &gpuResidlualStoresWrapper, sizeof(uint32_t*) * numStores));
    CUDA_CHECK_RETURN(hipMalloc((void**) &gpuPrefixStoresWrapper, sizeof(uint32_t*) * numStores));
    
// #pragma omp parallel for 
    for (uint32_t dataBlockIndex = 0; dataBlockIndex < numStores; ++dataBlockIndex) {
      uint32_t elementsInDataBlock = (((dataBlockIndex + 1)*chunkSize <= elementCount) ? chunkSize : chunkSize-((dataBlockIndex + 1)*chunkSize-elementCount));
      uint32_t sizeOfPrefixArray = (elementsInDataBlock * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity +
                                 ((elementsInDataBlock * gpuBitCountForRepresentation) % gpuStorageIndiceCapacity != 0);
      prefixStore[dataBlockIndex] = new uint32_t[sizeOfPrefixArray];
      prefixSizesStore[dataBlockIndex] = sizeOfPrefixArray; 
      residlualStore[dataBlockIndex] = new uint32_t[elementsInDataBlock+1]; //this padding actually waste less space than having a count array, +1 to avoid a branch later on when writing the remainder of the residuals
      chunkSizes[dataBlockIndex] = elementsInDataBlock;
      CUDA_CHECK_RETURN(hipMalloc((void**) &gpuPrefixMemoryStores[dataBlockIndex], sizeof(uint32_t) * sizeOfPrefixArray));
      CUDA_CHECK_RETURN(hipMemset(gpuPrefixMemoryStores[dataBlockIndex],0,sizeof(uint32_t) * sizeOfPrefixArray));
      CUDA_CHECK_RETURN(hipMalloc((void**) &gpuResidlualMemoryStores[dataBlockIndex], sizeof(uint32_t) * (elementsInDataBlock+1)));
      CUDA_CHECK_RETURN(hipMemset(gpuResidlualMemoryStores[dataBlockIndex],0,sizeof(uint32_t)*(elementsInDataBlock+1)));
    }
    //copy the alloced pointers to the wrapper array so we can send it off the kernel:
    CUDA_CHECK_RETURN(hipMemcpy(gpuResidlualStoresWrapper,gpuResidlualMemoryStores, 
				   sizeof(uint32_t*) * numStores, hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(gpuPrefixStoresWrapper,gpuPrefixMemoryStores, 
				   sizeof(uint32_t*) * numStores, hipMemcpyHostToDevice));
    timer::tic();
    gpuCompressionKernel<<<numStores, (chunkSize)/2, (chunkSize) * 4 * sizeof(uint32_t)>>>(gpuData,
			_gpuCompressorIV,elementCount,chunkSize,gpuPrefixStoresWrapper,
			gpuResidlualStoresWrapper,gpuResidualSizesStore);
    hipDeviceSynchronize();
    CUDA_CHECK_RETURN(hipGetLastError());
    _gpuCompressorAccumulatedTime += timer::toc();
    //get the residual arrays sizes
    CUDA_CHECK_RETURN(hipMemcpy(residualSizesStore,gpuResidualSizesStore,
				   sizeof(uint32_t) * numStores,hipMemcpyDeviceToHost));
    
    //Now do the callback and free all resources afterwards except the IV:
    for (uint32_t i = 0; i < numStores; ++i){      
      uint32_t elementsInDataBlock = (((i + 1)*chunkSize <= elementCount) ? chunkSize : chunkSize-((i + 1)*chunkSize-elementCount));
      uint32_t sizeOfPrefixArray = (elementsInDataBlock * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity +
                                 ((elementsInDataBlock * gpuBitCountForRepresentation) % gpuStorageIndiceCapacity != 0);		 
      CUDA_CHECK_RETURN(hipMemcpy(prefixStore[i],gpuPrefixMemoryStores[i], 
				   sizeof(uint32_t) * sizeOfPrefixArray, hipMemcpyDeviceToHost));
//       printf("COPYING %d bytes from graphics memory for block %d\n",residualSizesStore[i],i);
      CUDA_CHECK_RETURN(hipMemcpy(residlualStore[i],gpuResidlualMemoryStores[i], 
				   sizeof(uint32_t) * residualSizesStore[i], hipMemcpyDeviceToHost));
      _gpuAccumCompressedDataSize += residualSizesStore[i] + prefixSizesStore[i] + 1;
    }
    callBack(elementCount,residualSizesStore,residlualStore,prefixSizesStore,prefixStore,numStores,chunkSizes);
    for (uint32_t i = 0; i < numStores; ++i){
       CUDA_CHECK_RETURN(hipFree((void*) gpuResidlualMemoryStores[i]));
       CUDA_CHECK_RETURN(hipFree((void*) gpuPrefixMemoryStores[i]));
       delete[] residlualStore[i];
       delete[] prefixStore[i];
    }
    CUDA_CHECK_RETURN(hipFree((void*) gpuData));
    delete [] gpuResidlualMemoryStores;
    delete [] gpuPrefixMemoryStores;
    CUDA_CHECK_RETURN(hipFree((void*) gpuResidlualStoresWrapper));
    CUDA_CHECK_RETURN(hipFree((void*) gpuPrefixStoresWrapper));
    CUDA_CHECK_RETURN(hipFree((void*) gpuResidualSizesStore));
    delete[] residlualStore;
    delete[] prefixStore;
    delete[] residualSizesStore;
    delete[] prefixSizesStore;
    delete[] chunkSizes;
}

/*
 * Inits the decompressor
 * @params iv the first dataframe that serves as a basis for the decompresson of further dataframes
 * @params ivlength the length of the iv vector
 * @throws invalidInitializationException if the IV is empty
 */
void gpuCode::decompressor::initDecompressor(const float* iv, uint64_t ivLength){
  if (ivLength < 1)
    throw invalidInitializationException();
  if (_gpuDecompressorIV != NULL)
    delete[] _gpuDecompressorIV;
  _gpuDecompressorIV = new uint32_t[ivLength];
  memcpy(_gpuDecompressorIV,iv,ivLength*sizeof(float));
  _gpuDecompressorIVLength = ivLength;
  _gpuDecompressorAccumulatedTime = 0;
  _gpuAccumDecompressedDataSize = ivLength;
}

/*
 * Releases resources held by the decompressor
 */
void gpuCode::decompressor::releaseResources(){
  if (_gpuDecompressorIV != NULL){
    delete[] _gpuDecompressorIV;
    _gpuDecompressorIV = NULL;
    _gpuDecompressorIVLength = -1;
  }
}

/*
 * Gets the accumulated time since decompressor initialization
 */
double gpuCode::decompressor::getAccumulatedRunTimeSinceInit(){
  return _gpuDecompressorAccumulatedTime;
}

/*
 * Gets the accumulated size of the decompressed data since initialization 
 * (this can be used to compute the compression ratio)
 */
uint32_t gpuCode::decompressor::getAccumulatedDecompressedDataSize(){
  return _gpuAccumDecompressedDataSize;
}

/*
 * Decompresses a dataframe. This function will decompress a dataframe in parallel and will call back with the decompressed data when completed.
 * The user should save the initialization vector dataframe and the elementCount to file himself. For dataframe index > 1 the user
 * should save the decompressed frame to persistent storage within the scope of the callback function. After the
 * return of the callback function the decompressed data will be deleted from memory and the pointers will no longer be valid.
 * @throws invalidInitializationException if the length of the dataframe vector does not match the length of the initialization vector
 */
void gpuCode::decompressor::decompressData(uint32_t elementCount, uint32_t chunkCount, uint32_t * chunkSizes, 
			uint32_t ** compressedResiduals, uint32_t ** compressedPrefixes, 
			void (*callBack)(uint32_t elementCount, uint32_t * decompressedData)){
  if (_gpuDecompressorIV == NULL || _gpuDecompressorIVLength != elementCount)
        throw invalidInitializationException();
  timer::tic();
  #pragma omp parallel for 
  for (uint32_t dataBlockIndex = 0; dataBlockIndex < chunkCount; ++dataBlockIndex) {
    gpuDecompressionKernel(chunkSizes[0],chunkSizes[dataBlockIndex],compressedPrefixes[dataBlockIndex],
			compressedResiduals[dataBlockIndex],dataBlockIndex,dataBlockIndex*chunkSizes[0]);
  }
  _gpuDecompressorAccumulatedTime += timer::toc();
  callBack(elementCount, _gpuDecompressorIV);
  _gpuAccumDecompressedDataSize += elementCount;
}
