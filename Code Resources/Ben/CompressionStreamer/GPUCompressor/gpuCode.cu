#include "hip/hip_runtime.h"
/*
    Predictive 32-bit IEEE 754 floating point data compressor
    Copyright (C) 2013  benjamin bennahugo@aol.com

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "gpuCode.h"

uint32_t * _gpuCompressorIV = NULL;
uint64_t _gpuCompressorIVLength = -1;
uint32_t _gpuAccumCompressedDataSize = 0;
uint32_t * _gpuDecompressorIV = NULL;
uint64_t _gpuDecompressorIVLength = -1;
double _gpuCompressorAccumulatedTime = 0;
double _gpuDecompressorAccumulatedTime = 0;
uint32_t _gpuAccumDecompressedDataSize = 0;
const uint8_t gpuStorageIndiceCapacity = 8*sizeof(uint32_t);
const uint8_t gpuBitCountForRepresentation = 2;
uint32_t gpuBlockSize = 0; //set by initCUDA
/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }
/**
 * Initializes the device and sets up the block sizes and prints out memory properties
 * 
 */
void gpuCode::initCUDA(){
  int deviceCount, device;
    int gpuDeviceCount = 0;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t * properties = new hipDeviceProp_t[deviceCount];
    
    if (cudaResultCode != hipSuccess) 
        deviceCount = 0;
    /* machines with no GPUs can still report one emulation device */
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties[device], device);
        if (properties[device].major != 9999) /* 9999 means emulation only */
            ++gpuDeviceCount;
    }
    std::cout << gpuDeviceCount << " GPU CUDA device(s) found" << std::endl;
    if (gpuDeviceCount == 0){
	std::cout << "FATAL: NO CUDA CAPABLE CARDS ON THIS SYSTEM" << std::endl;
        exit(1);
    }
    hipSetDevice(0);
    gpuBlockSize = 256;
    size_t free = 0;
    size_t total = 0;
    hipMemGetInfo(&free,&total);
    std::cout << "Total GPU Memory on card: " << total/1024/1024 << " MB" << std::endl;
    std::cout << "Total GPU Memory available: " << free/1024/1024 << " MB" << std::endl;
    delete[] properties;
}
/**
 * Releases and resets GPU
 * 
 */
void gpuCode::releaseCard(){
  CUDA_CHECK_RETURN(hipDeviceReset());
}
/*
 * Inits the compressor
 * @params iv the first dataframe that serves as a basis for the compresson of further dataframes
 * @params ivlength the length of the iv vector
 * @throws invalidInitializationException if the IV is empty
 */
void gpuCode::compressor::initCompressor(const float* iv, uint64_t ivLength){
  if (ivLength < 1)
    throw invalidInitializationException();
  uint32_t numBlocks = (ivLength / gpuBlockSize) + (ivLength%gpuBlockSize != 0); //+1 iff there is remaining elements after number of completely fulled blocks
  if (_gpuCompressorIV != NULL)
    CUDA_CHECK_RETURN(hipFree(_gpuCompressorIV));
  CUDA_CHECK_RETURN(hipMalloc((void**) &_gpuCompressorIV, sizeof(uint32_t) * numBlocks * gpuBlockSize));
  CUDA_CHECK_RETURN(hipMemset(_gpuCompressorIV,0,sizeof(uint32_t) * numBlocks * gpuBlockSize)); //ensure padding is set to zero on the device
  CUDA_CHECK_RETURN(hipMemcpy(_gpuCompressorIV, iv, ivLength*sizeof(float), hipMemcpyHostToDevice));
  _gpuCompressorIVLength = ivLength;
  _gpuCompressorAccumulatedTime = 0;
  _gpuAccumCompressedDataSize = ivLength+1;
}

/*
 * Releases resources held by the compressor
 */
void gpuCode::compressor::releaseResources(){
  if (_gpuCompressorIV != NULL){
    CUDA_CHECK_RETURN(hipFree((void*) _gpuCompressorIV));
    _gpuCompressorIV = NULL;
    _gpuCompressorIVLength = -1;
  }
}

/*
 * Gets the accumulated time since compressor initialization
 */
double gpuCode::compressor::getAccumulatedRunTimeSinceInit(){
  return _gpuCompressorAccumulatedTime;
}

/*
 * Gets the accumulated size needed to store the data since initialization of the compressor 
 * (this can be used to compute the compression ratio)
 */
uint32_t gpuCode::compressor::getAccumulatedCompressedDataSize(){
  return _gpuAccumCompressedDataSize;
}

const uint32_t BYTESPERINTMIN1_2 = sizeof(uint32_t) * sizeof(uint8_t) - 1;
inline int32_t imax_2( int32_t x, int32_t y )
{
    return x - ((x - y) & ((x - y) >> (BYTESPERINTMIN1_2)));
}
inline int32_t imin_2( int32_t x, int32_t y )
{
    return y ^ ((x ^ y) & -(x < y)); // min(x, y)
}

__device__ void storePrefixStream(const uint32_t * iv, uint32_t elementCount, uint32_t chunkSize, 
			uint32_t * residualAndPrefixStore, 
			uint32_t lowerBound,uint32_t blockThreadId,uint32_t index,uint32_t element,
			uint32_t bankOffset, uint32_t prefixArrOffset, uint32_t numBlocks,uint32_t prefix){
    extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement  
    
    uint32_t lshiftAmountPrefixes = gpuStorageIndiceCapacity - gpuBitCountForRepresentation;
    if ((index < elementCount)){
        //save the prefixes:
        uint32_t iTimesgpuBitCountForRepresentation = blockThreadId*gpuBitCountForRepresentation;
        uint32_t startingIndex = (iTimesgpuBitCountForRepresentation) >> 5;
        uint32_t rshiftAmount = (iTimesgpuBitCountForRepresentation) % gpuStorageIndiceCapacity;
	atomicOr(residualAndPrefixStore + numBlocks + prefixArrOffset + blockIdx.x * ((chunkSize * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity + 1) + startingIndex,
		 ((prefix << lshiftAmountPrefixes) >> rshiftAmount)); //according to the cuda developer guide this will compute the or and store it back to the same address
    }
}

/**
 * Define a macro according to Nvidia's GPU Gems 3 to offset the indexing in order to avoid bank conflicts
 * Compute Capability 	1.x: 16 banks
 * 			2.x/3.x: 32 banks
 */
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(n) \ ((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(n) ((n) >> LOG_NUM_BANKS)
#endif

__device__ void computeScan(uint32_t n, uint32_t blockSize) {
	extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement
	int thid = threadIdx.x; //TODO::use a more complex indexing scheme to deal with larger arrays
	int blockOffset = blockIdx.x*blockSize;
	int offset = 1;
	//up-sweep:
	for (int d = blockSize>>1; d > 0; d >>= 1)                    // build sum in place up the tree
	{
		__syncthreads();
		if (thid < d){
			int thidTimes2 = (thid<<1);
		    int ai = offset*(thidTimes2+1)-1;
		    int bi = offset*(thidTimes2+2)-1;
		    ai += CONFLICT_FREE_OFFSET(ai);
		    bi += CONFLICT_FREE_OFFSET(bi);
		    counts[bi] += counts[ai];
		}
		offset <<= 1;
	}
	//clear:
	if (thid==0) {
		int iMax = blockSize - 1;
		counts[iMax + CONFLICT_FREE_OFFSET(iMax)] = 0;
	}
	//down-sweep:
	for (int d = 1; d < blockSize; d <<= 1) // traverse down tree & build scan
	{
		offset >>= 1;
		__syncthreads();
		if (thid < d){
			int thidTimes2 = (thid<<1);
			int ai = offset*(thidTimes2+1)-1;
			int bi = offset*(thidTimes2+2)-1;
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			uint32_t t = counts[ai];
			counts[ai] = counts[bi];
			counts[bi] += t;
		}
	}
}

__device__ uint32_t storeResidualStream(uint32_t elementCount, uint32_t chunkSize, 
			uint32_t * residualAndPrefixStore, 
			uint32_t lowerBound,uint32_t blockThreadId,uint32_t index,uint32_t xoredElement,
			uint32_t bankOffset, uint32_t numBlocks){
  extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement  
  if ((index < elementCount)){ //exclude the last element
	//save the residuals:
        uint32_t countIndexN = blockThreadId+bankOffset;
        uint32_t accumulatedIndex = counts[countIndexN];
	uint32_t count = counts[(chunkSize<<1) + countIndexN]; //get the original count value before the prefix scan was computed
        uint32_t startingIndex = accumulatedIndex >> 5;
        uint8_t lshiftAmount = (gpuStorageIndiceCapacity - count);
        uint32_t rshiftAmount = accumulatedIndex % gpuStorageIndiceCapacity;
        uint8_t writtenBits = gpuStorageIndiceCapacity - lshiftAmount - max(rshiftAmount-lshiftAmount,0);
        atomicOr(residualAndPrefixStore + numBlocks +  blockIdx.x * (chunkSize + 1) + startingIndex,((xoredElement << lshiftAmount) >> rshiftAmount));
        atomicOr(residualAndPrefixStore + numBlocks + blockIdx.x * (chunkSize + 1) + startingIndex + 1,(xoredElement << (lshiftAmount + writtenBits - 1) << 1));    
    }
}
__global__ void gpuCompressionKernel(const uint32_t * data, uint32_t * iv, uint32_t elementCount, uint32_t chunkSize, 
			uint32_t * residualAndPrefixStore, uint32_t prefixArrOffset, uint32_t numBlocks){
      extern __shared__ uint32_t counts[]; //the kernel must be called with "length" as a third special arguement
    
    //Create difference array, count used bits (up to 3 bytes of leading zeros) and save prefixes
    uint32_t lowerBound = blockIdx.x*chunkSize;
    uint32_t blockThreadId = threadIdx.x;
    //in line with the way GPU Gems 3 structures the parallel prefix sum we have to copy TWO data elements into registers
    uint32_t ai = blockThreadId;
    uint32_t bi = blockThreadId + (chunkSize>>1);
    uint32_t bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    uint32_t bankOffsetB = CONFLICT_FREE_OFFSET(bi);
    uint32_t  blockOffsetA = lowerBound + ai;
    uint32_t  blockOffsetB = lowerBound + bi;
    uint32_t dataElementA = NULL;
    uint32_t dataElementB = NULL;
    uint32_t lastElemVal = 0;
    uint32_t xoredElementA = 0;
    uint32_t xoredElementB = 0;
    uint32_t prefixA = 0;
    uint32_t prefixB = 0;
    if (blockOffsetA < elementCount){
      uint32_t ivElement = iv[blockOffsetA];
      dataElementA = data[blockOffsetA];
      xoredElementA = ivElement ^ dataElementA;
      prefixA = min(3,__clz(xoredElementA) >> 3);
      //store a copy of the orignal count at an 2* BLOCK SIZE offset in shared memory so that we can get the scan values and originals later!
      uint32_t countIndexN = ai+bankOffsetA;
      counts[countIndexN] = ((sizeof(uint32_t)-prefixA) << 3);
      counts[(chunkSize<<1)+countIndexN] = counts[countIndexN];
    }
    if (blockOffsetB < elementCount){
      uint32_t ivElement = iv[blockOffsetB];
      dataElementB = data[blockOffsetB];
      xoredElementB = ivElement ^ dataElementB;
      prefixB = min(3,__clz(xoredElementB) >> 3);
      //store a copy of the orignal count at an 2* BLOCK SIZE offset in shared memory so that we can get the scan values and originals later!
      uint32_t countIndexN = bi+bankOffsetB;
      counts[countIndexN] = ((sizeof(uint32_t)-prefixB) << 3);
      counts[(chunkSize<<1)+countIndexN] = counts[countIndexN];
    }
    __syncthreads();
    //compute lzc and save the prefixes:
     storePrefixStream(iv,elementCount,chunkSize,residualAndPrefixStore,lowerBound,ai,blockOffsetA,xoredElementA,bankOffsetA,prefixArrOffset,numBlocks,prefixA);
     storePrefixStream(iv,elementCount,chunkSize,residualAndPrefixStore,lowerBound,bi,blockOffsetB,xoredElementB,bankOffsetB,prefixArrOffset,numBlocks,prefixB);
    __syncthreads();
    //compute parallel prefix sum (this method taken from GPU GEMS 3 computes 2 elements at a time):
     computeScan(elementCount,chunkSize);
     __syncthreads();
    //now save the residuals:
    storeResidualStream(elementCount,chunkSize,residualAndPrefixStore,lowerBound,ai,blockOffsetA,xoredElementA,bankOffsetA,numBlocks);
    storeResidualStream(elementCount,chunkSize,residualAndPrefixStore,lowerBound,bi,blockOffsetB,xoredElementB,bankOffsetB,numBlocks);
    __syncthreads();
    //calculate storage space used by residuals:
    if (blockOffsetA == elementCount-1){ //last element before end of block
       uint32_t accumulatedIndex = counts[bankOffsetA+ai] + counts[(chunkSize << 1) + bankOffsetA+ai];
       uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
                           (accumulatedIndex % gpuStorageIndiceCapacity != 0);
        residualAndPrefixStore[blockIdx.x] = sizeOfResidualArray; 
    } else if (blockOffsetB == elementCount-1){
       uint32_t accumulatedIndex = counts[bankOffsetB+bi] + counts[(chunkSize << 1) + bankOffsetB+bi];
       uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
                           (accumulatedIndex % gpuStorageIndiceCapacity != 0);
        residualAndPrefixStore[blockIdx.x] = sizeOfResidualArray; 
    } else if (blockThreadId == (chunkSize >> 1)-1){ //last thread of block
       uint32_t lastCountElemIndex = bankOffsetB+bi;
      uint32_t accumulatedIndex = counts[lastCountElemIndex] + counts[(chunkSize << 1) + lastCountElemIndex];
      uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
                          (accumulatedIndex % gpuStorageIndiceCapacity != 0);
	residualAndPrefixStore[blockIdx.x] = sizeOfResidualArray; 
    }
    //Copy the current data to the IV memory for the next round of compression
    if (blockOffsetA < elementCount)
      iv[blockOffsetA] = data[blockOffsetA];
    if (blockOffsetB < elementCount)
      iv[blockOffsetB] = data[blockOffsetB];
}

void gpuDecompressionKernel(uint32_t chunkSize, uint32_t dataBlockSize, 
			  uint32_t * compressedPrefixes, uint32_t * compressedResiduals,
			  uint32_t dataBlockIndex,uint32_t lowerBound) {
    uint32_t accumulatedIndex = 0;
    uint8_t lshiftAmount = (gpuStorageIndiceCapacity - gpuBitCountForRepresentation);
    for (uint32_t i = 0; i < dataBlockSize; ++i) {
	//inflate prefix
	uint32_t prefixIndex = i*gpuBitCountForRepresentation;
        uint32_t startingIndex = prefixIndex >> 5;
        uint8_t rshiftAmount = prefixIndex % gpuStorageIndiceCapacity;
        uint8_t prefix = ((compressedPrefixes[startingIndex] << rshiftAmount) >> lshiftAmount);
        uint32_t count = gpuStorageIndiceCapacity - (prefix << 3);
	//inflate its associated residual
	startingIndex = accumulatedIndex >> 5;
        uint8_t residuallshiftAmount = (gpuStorageIndiceCapacity - count);
        rshiftAmount = accumulatedIndex % gpuStorageIndiceCapacity;
        uint8_t writtenBits = gpuStorageIndiceCapacity - residuallshiftAmount - imax_2(rshiftAmount-residuallshiftAmount,0);
        register uint32_t residual = ( (compressedResiduals[startingIndex] << rshiftAmount) >> residuallshiftAmount);
        residual |= 
	  ( compressedResiduals[startingIndex+(gpuStorageIndiceCapacity - residuallshiftAmount - writtenBits > 0)] >> (residuallshiftAmount + writtenBits - 1) >> 1);
        _gpuDecompressorIV[lowerBound+i] ^= residual;
        accumulatedIndex += count;
    }
}


/*
 * Compresses a dataframe. This function will compress a dataframe in parallel and will call back with the compressed data when completed.
 * The user should save the initialization vector dataframe and the elementCount to file himself. For dataframe index > 1 the user
 * should save the compressed prefix and residual array to persistent storage within the scope of the callback function. After the
 * return of the callback function the compressed data will be deleted from memory and the pointers will no longer be valid.
 * @throws invalidInitializationException if the length of the dataframe vector does not match the length of the initialization vector
 */

void gpuCode::compressor::compressData(const float * data, uint32_t elementCount,
			  void (*callBack)(uint32_t elementCount, uint32_t * compressedResidualsIntCounts, uint32_t ** compressedResiduals,
			    uint32_t * compressedPrefixIntCounts, uint32_t ** compressedPrefixes, uint32_t chunkCount, uint32_t * chunkSizes)){
    if (_gpuCompressorIV == NULL || _gpuCompressorIVLength != elementCount)
        throw invalidInitializationException();
    uint32_t chunkSize = gpuBlockSize; 
    uint32_t numStores = elementCount/gpuBlockSize + (elementCount%gpuBlockSize != 0); //+1 iff there is remaining elements after number of completely fulled blocks
    uint32_t** residlualStore = new uint32_t*[numStores];
    uint32_t** prefixStore = new uint32_t*[numStores];
    uint32_t* residualSizesStore = new uint32_t[numStores];
    uint32_t* prefixSizesStore = new uint32_t[numStores];
    uint32_t* chunkSizes = new uint32_t[numStores];
    //alloc space for the data on the card
    uint32_t* gpuData = NULL;
    CUDA_CHECK_RETURN(hipMalloc((void**) &gpuData, sizeof(uint32_t) * numStores * gpuBlockSize)); //pad so that we completely fill every SM
    CUDA_CHECK_RETURN(hipMemcpy(gpuData, data, elementCount*sizeof(float), hipMemcpyHostToDevice));
    timer::tic();
    //alloc space for residuals and prefixes on the card
    uint32_t* gpuResidualsAndPrefixesStore = NULL;
    uint32_t* gpuPrefixSizesStore = NULL;    
    //create the wrappers for the residual and prefix memory stores which will be sent to the kernel: 
    uint32_t sizeOfPrefixArray = sizeof(uint32_t)*((elementCount * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity + elementCount); //pad the prefix array with 1 for every block to prevent branch diversion in the kernel
    uint32_t sizeOfResidualArray = sizeof(uint32_t)*(chunkSize*numStores + elementCount); //pad the residual array with 1 to prevent branch diversion in the kernel
    uint32_t sizeOfResidualSizesArray = sizeof(uint32_t) * numStores;
    uint32_t prefixArrOffset = (chunkSize*numStores + elementCount);
    CUDA_CHECK_RETURN(hipMalloc((void**) &gpuResidualsAndPrefixesStore, sizeOfResidualArray+sizeOfPrefixArray+sizeOfResidualSizesArray));
    CUDA_CHECK_RETURN(hipMemset(gpuResidualsAndPrefixesStore,0,sizeOfPrefixArray+sizeOfResidualArray+sizeOfResidualSizesArray));
   
    gpuCompressionKernel<<<numStores, (chunkSize)/2, (chunkSize) * 4 * sizeof(uint32_t)>>>(gpuData,
			_gpuCompressorIV,elementCount,chunkSize,
			gpuResidualsAndPrefixesStore,prefixArrOffset,numStores);
    hipDeviceSynchronize();
    CUDA_CHECK_RETURN(hipGetLastError());
    //copy the padded prefix and residual arrays over so that it can be bundled into 2 transactions and not hundreds:
//     uint32_t * tempResidualsAndPrefixesStore = NULL;
//     CUDA_CHECK_RETURN(hipHostMalloc((void**)&tempResidualsAndPrefixesStore,sizeOfResidualArray+sizeOfPrefixArray+sizeOfResidualSizesArray,0));
    uint32_t * tempResidualsAndPrefixesStore = (uint32_t*) malloc(sizeOfResidualArray+sizeOfPrefixArray+sizeOfResidualSizesArray);
    CUDA_CHECK_RETURN(hipMemcpy(tempResidualsAndPrefixesStore,gpuResidualsAndPrefixesStore, 
				   sizeOfResidualArray+sizeOfPrefixArray+sizeOfResidualSizesArray, hipMemcpyDeviceToHost));
    _gpuCompressorAccumulatedTime += timer::toc();
    //Now split the cuda memory up into unpadded chunks:
    uint32_t offsetPrefixes = 0;
    uint32_t offsetResiduals = 0;
    memcpy(residualSizesStore,tempResidualsAndPrefixesStore,sizeOfResidualSizesArray);
    for (uint32_t i = 0; i < numStores; ++i){      
      uint32_t elementsInDataBlock = (((i + 1)*chunkSize <= elementCount) ? chunkSize : chunkSize-((i + 1)*chunkSize-elementCount));
      uint32_t sizeOfPrefixArray = (elementsInDataBlock * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity +
                                 ((elementsInDataBlock * gpuBitCountForRepresentation) % gpuStorageIndiceCapacity != 0);		 
      //  std::cout << "Copying store " << i+1 << " of " << numStores << ", with prefix array length "<< sizeOfPrefixArray <<" and residual array length " << residualSizesStore[i] << std::endl;
      prefixStore[i] = new uint32_t[sizeOfPrefixArray];
      residlualStore[i] = new uint32_t[residualSizesStore[i]];
      prefixSizesStore[i] = sizeOfPrefixArray;
      chunkSizes[i] = elementsInDataBlock;
      memcpy(prefixStore[i],tempResidualsAndPrefixesStore + numStores + prefixArrOffset + offsetPrefixes, sizeof(uint32_t) * sizeOfPrefixArray);
      memcpy(residlualStore[i],tempResidualsAndPrefixesStore + numStores + offsetResiduals, sizeof(uint32_t) * residualSizesStore[i]);
      offsetPrefixes += ((chunkSize * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity + 1);
      offsetResiduals += (chunkSize + 1);
      _gpuAccumCompressedDataSize += tempResidualsAndPrefixesStore[i] + prefixSizesStore[i] + 1;
    }
    //_gpuCompressorAccumulatedTime += timer::toc();
//     hipHostFree(tempResidualsAndPrefixesStore);
    free(tempResidualsAndPrefixesStore);
    callBack(elementCount,residualSizesStore,residlualStore,prefixSizesStore,prefixStore,numStores,chunkSizes);
    for (uint32_t i = 0; i < numStores; ++i){
       delete[] residlualStore[i];
       delete[] prefixStore[i];
    }
    CUDA_CHECK_RETURN(hipFree((void*) gpuData));
    CUDA_CHECK_RETURN(hipFree((void*) gpuResidualsAndPrefixesStore));
    delete[] residlualStore;
    delete[] prefixStore;
    delete[] residualSizesStore;
    delete[] prefixSizesStore;
    delete[] chunkSizes;
}

/*
 * Inits the decompressor
 * @params iv the first dataframe that serves as a basis for the decompresson of further dataframes
 * @params ivlength the length of the iv vector
 * @throws invalidInitializationException if the IV is empty
 */
void gpuCode::decompressor::initDecompressor(const float* iv, uint64_t ivLength){
  if (ivLength < 1)
    throw invalidInitializationException();
  if (_gpuDecompressorIV != NULL)
    delete[] _gpuDecompressorIV;
  _gpuDecompressorIV = new uint32_t[ivLength];
  memcpy(_gpuDecompressorIV,iv,ivLength*sizeof(float));
  _gpuDecompressorIVLength = ivLength;
  _gpuDecompressorAccumulatedTime = 0;
  _gpuAccumDecompressedDataSize = ivLength;
}

/*
 * Releases resources held by the decompressor
 */
void gpuCode::decompressor::releaseResources(){
  if (_gpuDecompressorIV != NULL){
    delete[] _gpuDecompressorIV;
    _gpuDecompressorIV = NULL;
    _gpuDecompressorIVLength = -1;
  }
}

/*
 * Gets the accumulated time since decompressor initialization
 */
double gpuCode::decompressor::getAccumulatedRunTimeSinceInit(){
  return _gpuDecompressorAccumulatedTime;
}

/*
 * Gets the accumulated size of the decompressed data since initialization 
 * (this can be used to compute the compression ratio)
 */
uint32_t gpuCode::decompressor::getAccumulatedDecompressedDataSize(){
  return _gpuAccumDecompressedDataSize;
}

/*
 * Decompresses a dataframe. This function will decompress a dataframe in parallel and will call back with the decompressed data when completed.
 * The user should save the initialization vector dataframe and the elementCount to file himself. For dataframe index > 1 the user
 * should save the decompressed frame to persistent storage within the scope of the callback function. After the
 * return of the callback function the decompressed data will be deleted from memory and the pointers will no longer be valid.
 * @throws invalidInitializationException if the length of the dataframe vector does not match the length of the initialization vector
 */
void gpuCode::decompressor::decompressData(uint32_t elementCount, uint32_t chunkCount, uint32_t * chunkSizes, 
			uint32_t ** compressedResiduals, uint32_t ** compressedPrefixes, 
			void (*callBack)(uint32_t elementCount, uint32_t * decompressedData)){
  if (_gpuDecompressorIV == NULL || _gpuDecompressorIVLength != elementCount)
        throw invalidInitializationException();
  timer::tic();
  #pragma omp parallel for 
  for (uint32_t dataBlockIndex = 0; dataBlockIndex < chunkCount; ++dataBlockIndex) {
    gpuDecompressionKernel(chunkSizes[0],chunkSizes[dataBlockIndex],compressedPrefixes[dataBlockIndex],
			compressedResiduals[dataBlockIndex],dataBlockIndex,dataBlockIndex*chunkSizes[0]);
  }
  _gpuDecompressorAccumulatedTime += timer::toc();
  callBack(elementCount, _gpuDecompressorIV);
  _gpuAccumDecompressedDataSize += elementCount;
}
