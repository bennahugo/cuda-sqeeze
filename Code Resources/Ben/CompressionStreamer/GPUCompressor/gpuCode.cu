#include "hip/hip_runtime.h"
/*
    Predictive 32-bit IEEE 754 floating point data compressor
    Copyright (C) 2013  benjamin bennahugo@aol.com

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "gpuCode.h"

uint32_t * _gpuCompressorIV = NULL;
uint64_t _gpuCompressorIVLength = -1;
uint32_t _gpuAccumCompressedDataSize = 0;
uint32_t * _gpuDecompressorIV = NULL;
uint64_t _gpuDecompressorIVLength = -1;
double _gpuCompressorAccumulatedTime = 0;
double _gpuDecompressorAccumulatedTime = 0;
uint32_t _gpuAccumDecompressedDataSize = 0;
const uint8_t gpuStorageIndiceCapacity = 8*sizeof(uint32_t);
const uint8_t gpuBitCountForRepresentation = 2;

void initCUDA(){
  int deviceCount, device;
    int gpuDeviceCount = 0;
    struct hipDeviceProp_t properties;
    hipError_t cudaResultCode = hipGetDeviceCount(&deviceCount);
    if (cudaResultCode != hipSuccess) 
        deviceCount = 0;
    /* machines with no GPUs can still report one emulation device */
    for (device = 0; device < deviceCount; ++device) {
        hipGetDeviceProperties(&properties, device);
        if (properties.major != 9999) /* 9999 means emulation only */
            ++gpuDeviceCount;
    }
    std::cout << gpuDeviceCount << " GPU CUDA device(s) found" << std::endl;

    /* don't just return the number of gpus, because other runtime cuda
       errors can also yield non-zero return values */
    if (gpuDeviceCount == 0){
	std::cout << "FATAL: NO CUDA CAPABLE CARDS ON THIS SYSTEM" << std::endl;
        exit(1);
    }
}
/*
 * Inits the compressor
 * @params iv the first dataframe that serves as a basis for the compresson of further dataframes
 * @params ivlength the length of the iv vector
 * @throws invalidInitializationException if the IV is empty
 */
void gpuCode::compressor::initCompressor(const float* iv, uint64_t ivLength){
  if (ivLength < 1)
    throw invalidInitializationException();
  if (_gpuCompressorIV != NULL)
    _gpuCompressorIV = new uint32_t[ivLength];
  _gpuCompressorIV = new uint32_t[ivLength];
  memcpy(_gpuCompressorIV,iv,ivLength*sizeof(float));
  _gpuCompressorIVLength = ivLength;
  _gpuCompressorAccumulatedTime = 0;
  _gpuAccumCompressedDataSize = ivLength+1;
}

/*
 * Releases resources held by the compressor
 */
void gpuCode::compressor::releaseResources(){
  if (_gpuCompressorIV != NULL){
    free(_gpuCompressorIV);
    _gpuCompressorIV = NULL;
    _gpuCompressorIVLength = -1;
  }
}

/*
 * Gets the accumulated time since compressor initialization
 */
double gpuCode::compressor::getAccumulatedRunTimeSinceInit(){
  return _gpuCompressorAccumulatedTime;
}

/*
 * Gets the accumulated size needed to store the data since initialization of the compressor 
 * (this can be used to compute the compression ratio)
 */
uint32_t gpuCode::compressor::getAccumulatedCompressedDataSize(){
  return _gpuAccumCompressedDataSize;
}

void gpuCompressionKernel(const float * data, uint32_t elementCount, uint32_t dataBlockIndex, 
			uint32_t chunkSize, uint32_t ** prefixStore, uint32_t ** residualStore, 
			uint32_t * prefixSizeStore, uint32_t * residualSizeStore, uint32_t * dataBlockSizes,
			uint32_t lowerBound, uint32_t elementsInDataBlock){

//     //create storage for counts and prefixes:
//     uint32_t sizeOfPrefixArray = (elementsInDataBlock * gpuBitCountForRepresentation) / gpuStorageIndiceCapacity +
//                                  ((elementsInDataBlock * gpuBitCountForRepresentation) % gpuStorageIndiceCapacity != 0);
//     uint32_t * arrPrefix = new uint32_t[sizeOfPrefixArray];
//     memset(arrPrefix,0,sizeof(uint32_t)*sizeOfPrefixArray);
//     uint32_t * arrResiduals = new uint32_t[elementsInDataBlock+1]; //this padding actually waste less space than having a count array, +1 to avoid a branch later on when writing the remainder of the residuals
//     memset(arrResiduals,0,sizeof(uint32_t)*elementsInDataBlock);
// 
//     //Create difference array, count used bits (up to 3 bytes of leading zeros) and save prefixes
//     uint32_t lshiftAmountPrefixes = gpuStorageIndiceCapacity - gpuBitCountForRepresentation;
//     uint32_t accumulatedIndex = 0;
//     for (uint32_t i = 0; i < elementsInDataBlock; ++i) {
// 	uint32_t index = i+lowerBound;
//         //save the prefixes:
//         uint32_t element = (_gpuCompressorIV[index] ^= ((uint32_t*)&(data[0]))[index]);
// 	uint32_t prefix0 = imin(3,lzc (element) >> 3);
//         uint32_t iTimesgpuBitCountForRepresentation = i*gpuBitCountForRepresentation;
//         uint32_t startingIndex = (iTimesgpuBitCountForRepresentation) >> 5;
//         uint32_t rshiftAmount = (iTimesgpuBitCountForRepresentation) % gpuStorageIndiceCapacity;
//         arrPrefix[startingIndex] |= ((prefix0 << lshiftAmountPrefixes) >> rshiftAmount);
//         uint32_t count = ((sizeof(uint32_t)-prefix0) << 3);
//         
// 	//save the residuals:
//         startingIndex = accumulatedIndex >> 5;
//         uint8_t lshiftAmount = (gpuStorageIndiceCapacity - count);
//         rshiftAmount = accumulatedIndex % gpuStorageIndiceCapacity;
//         uint8_t writtenBits = gpuStorageIndiceCapacity - lshiftAmount - imax(rshiftAmount-lshiftAmount,0);
//         element = _gpuCompressorIV[index]; //it seems after _lzcnt_u32 touches a memory location it is not optimized correctly this is a work arround
//         arrResiduals[startingIndex] |= ( (element << lshiftAmount) >> rshiftAmount);
//         arrResiduals[startingIndex+1] |= (element << (lshiftAmount + writtenBits - 1) << 1);
//         accumulatedIndex += count;
//     }
//  
//     //calculate storage space used by residuals:
//     uint32_t sizeOfResidualArray = accumulatedIndex / gpuStorageIndiceCapacity +
//                           (accumulatedIndex % gpuStorageIndiceCapacity != 0);
// 
//     //Store pointers to the current prefixes and residuals
//     residualStore[dataBlockIndex] = arrResiduals;
//     prefixStore[dataBlockIndex] = arrPrefix;
//     residualSizeStore[dataBlockIndex] = sizeOfResidualArray;
//     prefixSizeStore[dataBlockIndex] = sizeOfPrefixArray; 
//     dataBlockSizes[dataBlockIndex] = elementsInDataBlock;
//     
//     
//     //Copy the current data to the IV memory for the next round of compression
//      memcpy(_gpuCompressorIV+lowerBound,data+lowerBound,elementsInDataBlock*sizeof(float));
//     //the prefixes and residluals will be freed by the caller
}


void gpuDecompressionKernel(uint32_t chunkSize, uint32_t dataBlockSize, 
			  uint32_t * compressedPrefixes, uint32_t * compressedResiduals,
			  uint32_t dataBlockIndex,uint32_t lowerBound) {
//     uint32_t accumulatedIndex = 0;
//     uint8_t lshiftAmount = (gpuStorageIndiceCapacity - gpuBitCountForRepresentation);
//     for (uint32_t i = 0; i < dataBlockSize; ++i) {
// 	//inflate prefix
// 	uint32_t prefixIndex = i*gpuBitCountForRepresentation;
//         uint32_t startingIndex = prefixIndex >> 5;
//         uint8_t rshiftAmount = prefixIndex % gpuStorageIndiceCapacity;
//         uint8_t prefix = ((compressedPrefixes[startingIndex] << rshiftAmount) >> lshiftAmount);
//         uint32_t count = gpuStorageIndiceCapacity - (prefix << 3);
// 	//inflate its associated residual
// 	startingIndex = accumulatedIndex >> 5;
//         uint8_t residuallshiftAmount = (gpuStorageIndiceCapacity - count);
//         rshiftAmount = accumulatedIndex % gpuStorageIndiceCapacity;
//         uint8_t writtenBits = gpuStorageIndiceCapacity - residuallshiftAmount - imax(rshiftAmount-residuallshiftAmount,0);
//         register uint32_t residual = ( (compressedResiduals[startingIndex] << rshiftAmount) >> residuallshiftAmount);
//         residual |= 
// 	  ( compressedResiduals[startingIndex+(gpuStorageIndiceCapacity - residuallshiftAmount - writtenBits > 0)] >> (residuallshiftAmount + writtenBits - 1) >> 1);
//         _gpuDecompressorIV[lowerBound+i] ^= residual;
//         accumulatedIndex += count;
//     }
}


/*
 * Compresses a dataframe. This function will compress a dataframe in parallel and will call back with the compressed data when completed.
 * The user should save the initialization vector dataframe and the elementCount to file himself. For dataframe index > 1 the user
 * should save the compressed prefix and residual array to persistent storage within the scope of the callback function. After the
 * return of the callback function the compressed data will be deleted from memory and the pointers will no longer be valid.
 * @throws invalidInitializationException if the length of the dataframe vector does not match the length of the initialization vector
 */

void gpuCode::compressor::compressData(const float * data, uint32_t elementCount,
			  void (*callBack)(uint32_t elementCount, uint32_t * compressedResidualsIntCounts, uint32_t ** compressedResiduals,
			    uint32_t * compressedPrefixIntCounts, uint32_t ** compressedPrefixes, uint32_t chunkCount, uint32_t * chunkSizes)){
    if (_gpuCompressorIV == NULL || _gpuCompressorIVLength != elementCount)
        throw invalidInitializationException();
    timer::tic();
    uint32_t NUMTHREADS = omp_get_max_threads();
    uint32_t chunkSize = elementCount/NUMTHREADS; 
    uint32_t numStores = NUMTHREADS+(elementCount%NUMTHREADS != 0);
    uint32_t** residlualStore = new uint32_t*[numStores];
    uint32_t** prefixStore = new uint32_t*[numStores];
    uint32_t* residualSizesStore = new uint32_t[numStores];
    uint32_t* prefixSizesStore = new uint32_t[numStores];
    uint32_t* chunkSizes = new uint32_t[numStores];
#pragma omp parallel for 
    for (uint32_t dataBlockIndex = 0; dataBlockIndex < NUMTHREADS; ++dataBlockIndex) {  
      gpuCompressionKernel(data,elementCount,dataBlockIndex,chunkSize,prefixStore,
			residlualStore,prefixSizesStore,residualSizesStore,chunkSizes,
			dataBlockIndex*chunkSize,
			(((dataBlockIndex + 1)*chunkSize <= elementCount) ? chunkSize : chunkSize-((dataBlockIndex + 1)*chunkSize-elementCount)));
    }
      gpuCompressionKernel(data,elementCount,NUMTHREADS,chunkSize,prefixStore,
			residlualStore,prefixSizesStore,residualSizesStore,chunkSizes,
			NUMTHREADS*chunkSize,
			(((NUMTHREADS + 1)*chunkSize <= elementCount) ? chunkSize : chunkSize-((NUMTHREADS + 1)*chunkSize-elementCount)));
    _gpuCompressorAccumulatedTime += timer::toc();
    //Now do the callback and free all resources afterwards except the IV:
    for (uint32_t i = 0; i < numStores; ++i){
      _gpuAccumCompressedDataSize += residualSizesStore[i] + prefixSizesStore[i] + 1;
    }
    callBack(elementCount,residualSizesStore,residlualStore,prefixSizesStore,prefixStore,numStores,chunkSizes);
    for (uint32_t i = 0; i < numStores; ++i){
       free(residlualStore[i]);
       free(prefixStore[i]);
    }
    delete[] residlualStore;
    delete[] prefixStore;
    delete[] residualSizesStore;
    delete[] prefixSizesStore;
    delete[] chunkSizes;
}

/*
 * Inits the decompressor
 * @params iv the first dataframe that serves as a basis for the decompresson of further dataframes
 * @params ivlength the length of the iv vector
 * @throws invalidInitializationException if the IV is empty
 */
void gpuCode::decompressor::initDecompressor(const float* iv, uint64_t ivLength){
  if (ivLength < 1)
    throw invalidInitializationException();
  if (_gpuDecompressorIV != NULL)
    delete[] _gpuDecompressorIV;
  _gpuDecompressorIV = new uint32_t[ivLength];
  memcpy(_gpuDecompressorIV,iv,ivLength*sizeof(float));
  _gpuDecompressorIVLength = ivLength;
  _gpuDecompressorAccumulatedTime = 0;
  _gpuAccumDecompressedDataSize = ivLength;
}

/*
 * Releases resources held by the decompressor
 */
void gpuCode::decompressor::releaseResources(){
  if (_gpuDecompressorIV != NULL){
    delete[] _gpuDecompressorIV;
    _gpuDecompressorIV = NULL;
    _gpuDecompressorIVLength = -1;
  }
}

/*
 * Gets the accumulated time since decompressor initialization
 */
double gpuCode::decompressor::getAccumulatedRunTimeSinceInit(){
  return _gpuDecompressorAccumulatedTime;
}

/*
 * Gets the accumulated size of the decompressed data since initialization 
 * (this can be used to compute the compression ratio)
 */
uint32_t gpuCode::decompressor::getAccumulatedDecompressedDataSize(){
  return _gpuAccumDecompressedDataSize;
}

/*
 * Decompresses a dataframe. This function will decompress a dataframe in parallel and will call back with the decompressed data when completed.
 * The user should save the initialization vector dataframe and the elementCount to file himself. For dataframe index > 1 the user
 * should save the decompressed frame to persistent storage within the scope of the callback function. After the
 * return of the callback function the decompressed data will be deleted from memory and the pointers will no longer be valid.
 * @throws invalidInitializationException if the length of the dataframe vector does not match the length of the initialization vector
 */
void gpuCode::decompressor::decompressData(uint32_t elementCount, uint32_t chunkCount, uint32_t * chunkSizes, 
			uint32_t ** compressedResiduals, uint32_t ** compressedPrefixes, 
			void (*callBack)(uint32_t elementCount, uint32_t * decompressedData)){
  if (_gpuDecompressorIV == NULL || _gpuDecompressorIVLength != elementCount)
        throw invalidInitializationException();
  timer::tic();
  #pragma omp parallel for 
  for (uint32_t dataBlockIndex = 0; dataBlockIndex < chunkCount; ++dataBlockIndex) {
    gpuDecompressionKernel(chunkSizes[0],chunkSizes[dataBlockIndex],compressedPrefixes[dataBlockIndex],
			compressedResiduals[dataBlockIndex],dataBlockIndex,dataBlockIndex*chunkSizes[0]);
  }
  _gpuDecompressorAccumulatedTime += timer::toc();
  callBack(elementCount, _gpuDecompressorIV);
  _gpuAccumDecompressedDataSize += elementCount;
}
